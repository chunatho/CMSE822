/* 
 * Solves the Panfilov model using an explicit numerical scheme.
 * Based on code orginally provided by Xing Cai, Simula Research Laboratory 
 * and reimplementation by Scott B. Baden, UCSD
 * 
 * Modified and  restructured by Didem Unat, Koc University
 * Modified for GPU by Thomas Chuna and Aaron Magilligan
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>
#include <stdlib.h>
#include <iostream>
#include <iomanip>
#include <string.h>
#include <math.h>
#include <sys/time.h>
using namespace std;

# define IMAT(j,i) ( (i) + (n+2)*(j) )

// Utilities

// Timer: Make successive calls and take a difference to get the elapsed time.
static const double kMicro = 1.0e-6;
double getTime() {
	struct timeval TV;
    struct timezone TZ;

    const int RC = gettimeofday(&TV, &TZ);
    if (RC == -1) {
            cerr << "ERROR: Bad call to gettimeofday" << endl;
            return(-1);
    }

    return ( ( (double) TV.tv_sec ) + kMicro*( (double) TV.tv_usec) );

}  // end getTime()
    
// Reports statistics about the computation
// These values should not vary (except to within roundoff)
// when we use different numbers of  processes to solve the problem
double stats(double *E, int n, double *_mx) {
	double mx = -1;
	double l2norm = 0;

	for ( int j = 1; j <= n; j++ ) 
		for ( int i = 1; i <= n; i++ ) {

			l2norm += E[ IMAT(j,i) ]*E[ IMAT(j,i) ];
			if ( E[ IMAT(j,i) ] > mx )
				mx = E[ IMAT(j,i) ];

		}	

	*_mx = mx;
	l2norm /= ( (double) (n*n) );
	l2norm = sqrt(l2norm);

	return l2norm;
 }

// External functions
extern "C" {
    void splot(double *E, double T, int niter, int m, int n);
}
void cmdLine(int argc, char *argv[], double& T, int& n, int& px, int& py, int& plot_freq, int& no_comm, int&num_threads);

__global__
void kernel(double* E, double *E_prev, double* R, const int n, const double alpha, const double kk, 
		const double dt, const double a, const double epsilon, 
        	const double M1, const double M2, const double b)
{
	//kernel #1 to enforce boundary conditions

	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if(index <= n){
		E_prev[ IMAT(index,0) ] = E_prev[ IMAT(index,2) ];
		E_prev[ IMAT(index,n+1) ] = E_prev[ IMAT(index,n-1) ];
		E_prev[ IMAT(0,index) ] = E_prev[ IMAT(2,index) ];
		E_prev[ IMAT(n+1,index) ] = E_prev[ IMAT(n-1,index) ];
	}
	

	int i =  blockIdx.y*blockDim.y + threadIdx.y;
	int j =  blockIdx.x*blockDim.x + threadIdx.x;
	float tmpE = E[ IMAT(j,i) ];

	__syncthreads();
	//kernel #2 to solve PDE
	if(i<=n && j <=n){
		tmpE = E_prev[ IMAT(j,i) ]+alpha*(E_prev[ IMAT(j,i+1) ]
			  + E_prev[ IMAT(j,i-1) ]-4*E_prev[ IMAT(j,i) ]
			  + E_prev[ IMAT(j+1,i) ]+E_prev[ IMAT(j-1,i) ]);
	}

	__syncthreads();
	if(i<=n && j <=n){
	//kernel #3 to solve E ODE
		float tmpR = R[ IMAT(j,i) ];
		tmpE = tmpE - dt*(kk*tmpE*( tmpE - a )*( tmpE - 1 ) + tmpE*tmpR );
	//kernel #4 to solve R ODE
		R[ IMAT(j,i) ] = tmpR + dt*( epsilon + M1*tmpR / ( tmpE + M2) )*( -tmpR - kk*tmpE*(tmpE - b - 1 ) );
		E[ IMAT(j,i) ] = tmpE;
	}
}

// Main program
int main (int argc, char** argv) {

	/*
	*  Solution arrays
	*   E is the "Excitation" variable, a voltage
	*   R is the "Recovery" variable
	*   E_prev_old is the Excitation variable for the previous timestep,
	*      and is used in time integration
	*/

	// Various constants - these definitions shouldn't change
	const double a = 0.1, b = 0.1, kk = 8.0, M1 = 0.07, M2 = 0.3, epsilon = 0.01, d = 5e-5;

	double T = 1000.0;
	int n = 200;
	int plot_freq = 0;
	int px = 1, py = 1;
	int no_comm = 0;
	int num_threads=1;

	cmdLine( argc, argv, T, n,px, py, plot_freq, no_comm, num_threads);
    int array_size = sizeof(double)*(n+2)*(n+2);

	// Allocate contiguous memory for solution arrays
	// The computational box is defined on [1:m+1,1:n+1]
	// We pad the arrays in order to facilitate differencing on the 
	// boundaries of the computation box

	double *E, *R, *E_prev;
	E = (double*)malloc(array_size);
	E_prev = (double*)malloc(array_size);
	R = (double*)malloc(array_size);


	// Initialization
	for ( int j = 1; j <= n; j++ )
		for ( int i = 1; i <= n; i++ )
			E_prev[ IMAT(j,i) ] = R[ IMAT(j,i) ] = 0;

	for ( int j = 1; j <= n; j++ )
	        for ( int i = n/2 + 1; i <= n; i++ )
			E_prev[ IMAT(j,i) ] = 1.0;

	for ( int j = n/2 + 1; j <= n; j++ )
		for ( int i = 1; i <= n; i++ )
			R[ IMAT(j,i) ] = 1.0;

    

	double dx = 1.0/n;

	// For time integration, these values shouldn't change 
	double rp= kk*( b + 1 )*( b + 1 ) / 4;
	double dte = ( dx*dx ) / ( d*4 +  ( dx*dx ) * ( rp + kk ) );
	double dtr = 1 / ( epsilon + (M1/M2)*rp );
	double dt = ( dte < dtr ) ? 0.95*dte : 0.95* dtr;
	double alpha = d*dt / ( dx*dx );

	cout << "Grid Size       : " << n << endl; 
	cout << "Duration of Sim : " << T << endl; 
	cout << "Time step dt    : " << dt << endl; 
	cout << "Process geometry: " << px << " x " << py << endl;
	if (no_comm)
		cout << "Communication   : DISABLED" << endl;

	cout << endl;

	// Start the timer
	double t0 = getTime();


	// Simulated time is different from the integer timestep number

	// Simulated time
	double t = 0.0;

	// Integer timestep number
	int niter=0;
    
	//Initialize and Allocate GPU Memory
	double *d_E, *d_R, *d_E_prev;
	hipMalloc((void**)&d_E, array_size);
	hipMalloc((void**)&d_E_prev, array_size);
	hipMalloc((void**)&d_R, array_size);
	//Copy to CUDA kernels
	hipMemcpy(d_E, E, array_size, hipMemcpyHostToDevice);
	hipMemcpy(d_E_prev, E_prev, array_size, hipMemcpyHostToDevice);
	hipMemcpy(d_R, R, array_size, hipMemcpyHostToDevice);

	//Define Grid and block sizes
	dim3 DimGrid(ceil((n+2)/16.0), ceil((n+2)/16.0), 1);
	dim3 DimBlock(16, 16, 1);

	while ( t < T ) {
	
		t += dt;
		niter++;
		kernel<<<DimGrid,DimBlock>>>(d_E, d_E_prev, d_R,n,alpha,kk,dt,a,epsilon, M1, M2, b);

		//swap current E with previous E
		double *tmp = d_E; d_E = d_E_prev; d_E_prev = tmp;

		if ( plot_freq ) {
			int k = ( (int) (t / plot_freq) );
			if ( (t - k*plot_freq ) < dt ) {
		                hipMemcpy(E, d_E, array_size, hipMemcpyDeviceToHost);
				splot(E,t,niter,n+2,n+2);
			}
		}
	} //end of while loop

	//Retrieve Result and Clean up GPU memory
	hipMemcpy(E, d_E, array_size, hipMemcpyDeviceToHost);
	hipMemcpy(E_prev, d_E_prev, array_size, hipMemcpyDeviceToHost);
	hipMemcpy(R, d_R, array_size, hipMemcpyDeviceToHost);
	hipFree(d_E);
	hipFree(d_E_prev);
	hipFree(d_R);


	//Run Diagnostics
	double time_elapsed = getTime() - t0;
//	double Gflops = (double)(niter * (1E-9 * n * n ) * 28.0) / time_elapsed;
//	double BW = (double)(niter * 1E-9 * (n * n * sizeof(double) * 4.0  ))/time_elapsed;
	double mx;
	double l2norm = stats(E_prev,n,&mx);

        //Output Diagnostics
/*
  	cout << "Number of Iterations        : " << niter << endl;
        cout << "Elapsed Time (sec)          : " << time_elapsed << endl;
        cout << "Sustained Gflops Rate       : " << Gflops << endl;
        cout << "Sustained Bandwidth (GB/sec): " << BW << endl << endl;
        cout << "Max: " << mx <<  " L2norm: "<< l2norm << endl;
*/
  	printf("Niter: %d  time: %f L2norm: %f max: %f \n",niter,time_elapsed,l2norm,mx);



	if (plot_freq)	{	
		cout << "\n\nEnter any input to close the program and the plot..." << endl;
		getchar();
	}

	//Clean Up CPU memory and close up shop
	free (E);
	free (E_prev);
	free (R);
    
    
	return 0;
}
