/* 
 * Solves the Panfilov model using an explicit numerical scheme.
 * Based on code orginally provided by Xing Cai, Simula Research Laboratory 
 * and reimplementation by Scott B. Baden, UCSD
 * 
 * Modified and  restructured by Didem Unat, Koc University
 * Modified for GPU by Thomas Chuna and Aaron Magilligan
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>
#include <stdlib.h>
#include <iostream>
#include <iomanip>
#include <string.h>
#include <math.h>
#include <sys/time.h>
using namespace std;

# define IMAT(j,i) ( (i) + (n+2)*(j) )

// Utilities

// Timer: Make successive calls and take a difference to get the elapsed time.
static const double kMicro = 1.0e-6;
double getTime() {
	struct timeval TV;
    struct timezone TZ;

    const int RC = gettimeofday(&TV, &TZ);
    if (RC == -1) {
            cerr << "ERROR: Bad call to gettimeofday" << endl;
            return(-1);
    }

    return ( ( (double) TV.tv_sec ) + kMicro*( (double) TV.tv_usec) );

}  // end getTime()
    
// Reports statistics about the computation
// These values should not vary (except to within roundoff)
// when we use different numbers of  processes to solve the problem
double stats(double *E, int n, double *_mx) {
	double mx = -1;
	double l2norm = 0;

	for ( int j = 1; j <= n; j++ ) 
		for ( int i = 1; i <= n; i++ ) {

			l2norm += E[ IMAT(j,i) ]*E[ IMAT(j,i) ];
			if ( E[ IMAT(j,i) ] > mx )
				mx = E[ IMAT(j,i) ];

		}	

	*_mx = mx;
	l2norm /= ( (double) (n*n) );
	l2norm = sqrt(l2norm);

	return l2norm;
 }

// External functions
extern "C" {
    void splot(double *E, double T, int niter, int m, int n);
}
void cmdLine(int argc, char *argv[], double& T, int& n, int& px, int& py, int& plot_freq, int& no_comm, int&num_threads);

__global__
void kernel(double* E, double* R, const int n, const double alpha, const double kk, 
		const double dt, const double a, const double epsilon, 
        	const double M1, const double M2, const double b)
{
	//kernel #1 to enforce boundary conditions

	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if(index <= n){
		E[ IMAT(index,0) ] = E[ IMAT(index,2) ];
		E[ IMAT(index,n+1) ] = E[ IMAT(index,n-1) ];
		E[ IMAT(0,index) ] = E[ IMAT(2,index) ];
		E[ IMAT(n+1,index) ] = E[ IMAT(n-1,index) ];
	}
	

	int i =  blockIdx.y*blockDim.y + threadIdx.y;
	int j =  blockIdx.x*blockDim.x + threadIdx.x;
	double tmpR =R[ IMAT(j,i) ]; double tmpR2 =0.0;
	double tmpE =0.0; double tmpE2 =0.0;

	__syncthreads();
	//kernel #2 to solve PDE
	if(i<=n && j <=n){
		tmpE = E[ IMAT(j,i) ]+alpha*(E[ IMAT(j,i+1) ]
			  + E[ IMAT(j,i-1) ]-4*E[ IMAT(j,i) ]
			  + E[ IMAT(j+1,i) ]+E[ IMAT(j-1,i) ]);
	}

	__syncthreads();
	//kernel #3 to solve E ODE
	if(i<=n && j <=n)
		tmpE2 = tmpE - dt*(kk*tmpE*( tmpE - a )*( tmpE - 1 ) + tmpE*tmpR );

	__syncthreads();
	//kernel #4 to solve R ODE
	if(i<=n && j <=n)
		tmpR2 = tmpR + dt*( epsilon + M1*tmpR / ( tmpE2 + M2) )*( -tmpR - kk*tmpE2*(tmpE2 - b - 1 ) );

	E[ IMAT(j,i) ] = tmpE2;
	R[ IMAT(j,i) ] = tmpR2;

}

// Main program
int main (int argc, char** argv) {

	/*
	*  Solution arrays
	*   E is the "Excitation" variable, a voltage
	*   R is the "Recovery" variable
	*   E_prev_old is the Excitation variable for the previous timestep,
	*      and is used in time integration
	*/

	// Various constants - these definitions shouldn't change
	const double a = 0.1, b = 0.1, kk = 8.0, M1 = 0.07, M2 = 0.3, epsilon = 0.01, d = 5e-5;

	double T = 1000.0;
	int n = 200;
	int plot_freq = 0;
	int px = 1, py = 1;
	int no_comm = 0;
	int num_threads=1;

	cmdLine( argc, argv, T, n,px, py, plot_freq, no_comm, num_threads);
    int array_size = sizeof(double)*(n+2)*(n+2);

	// Allocate contiguous memory for solution arrays
	// The computational box is defined on [1:m+1,1:n+1]
	// We pad the arrays in order to facilitate differencing on the 
	// boundaries of the computation box

	double *E, *R;
	E = (double*)malloc(array_size);
	R = (double*)malloc(array_size);


	// Initialization
	for ( int j = 1; j <= n; j++ )
		for ( int i = 1; i <= n; i++ )
			E[ IMAT(j,i) ] = R[ IMAT(j,i) ] = 0;

	for ( int j = 1; j <= n; j++ )
	        for ( int i = n/2 + 1; i <= n; i++ )
			E[ IMAT(j,i) ] = 1.0;

	for ( int j = n/2 + 1; j <= n; j++ )
		for ( int i = 1; i <= n; i++ )
			R[ IMAT(j,i) ] = 1.0;

    

	double dx = 1.0/n;

	// For time integration, these values shouldn't change 
	double rp= kk*( b + 1 )*( b + 1 ) / 4;
	double dte = ( dx*dx ) / ( d*4 +  ( dx*dx ) * ( rp + kk ) );
	double dtr = 1 / ( epsilon + (M1/M2)*rp );
	double dt = ( dte < dtr ) ? 0.95*dte : 0.95* dtr;
	double alpha = d*dt / ( dx*dx );

	cout << "Grid Size       : " << n << endl; 
	cout << "Duration of Sim : " << T << endl; 
	cout << "Time step dt    : " << dt << endl; 
	cout << "Process geometry: " << px << " x " << py << endl;
	if (no_comm)
		cout << "Communication   : DISABLED" << endl;

	cout << endl;

	// Start the timer
	double t0 = getTime();


	// Simulated time is different from the integer timestep number

	// Simulated time
	double t = 0.0;

	// Integer timestep number
	int niter=0;
    
	//Initialize and Allocate GPU Memory
	double *d_E, *d_R;
	hipMalloc((void**)&d_E, array_size);
	hipMalloc((void**)&d_R, array_size);
	//Copy to CUDA kernels
	hipMemcpy(d_E, E, array_size, hipMemcpyHostToDevice);
	hipMemcpy(d_R, R, array_size, hipMemcpyHostToDevice);

	//Define Grid and block sizes
	dim3 DimGrid(ceil((n+2)/16.0), ceil((n+2)/16.0), 1);
	dim3 DimBlock(16, 16, 1);

	while ( t < T ) {
	
		t += dt;
		niter++;
		kernel<<<DimGrid,DimBlock>>>(d_E, d_R,n,alpha,kk,dt,a,epsilon, M1, M2, b);

		if ( plot_freq ) {
			int k = ( (int) (t / plot_freq) );
			if ( (t - k*plot_freq ) < dt ) {
		                hipMemcpy(E, d_E, array_size, hipMemcpyDeviceToHost);
				splot(E,t,niter,n+2,n+2);
			}
		}
	} //end of while loop

	//Retrieve Result and Clean up GPU memory
	hipMemcpy(E, d_E, array_size, hipMemcpyDeviceToHost);
	hipMemcpy(R, d_R, array_size, hipMemcpyDeviceToHost);
	hipFree(d_E);
	hipFree(d_R);


	//Run Diagnostics
	double time_elapsed = getTime() - t0;
	double Gflops = (double)(niter * (1E-9 * n * n ) * 28.0) / time_elapsed;
	double BW = (double)(niter * 1E-9 * (n * n * sizeof(double) * 4.0  ))/time_elapsed;
	double mx;
	double l2norm = stats(E,n,&mx);

	//Output Diagnostics
	cout << "Number of Iterations        : " << niter << endl;
	cout << "Elapsed Time (sec)          : " << time_elapsed << endl;
	cout << "Sustained Gflops Rate       : " << Gflops << endl; 
	cout << "Sustained Bandwidth (GB/sec): " << BW << endl << endl; 
	cout << "Max: " << mx <<  " L2norm: "<< l2norm << endl;

	if (plot_freq)	{	
		cout << "\n\nEnter any input to close the program and the plot..." << endl;
		getchar();
	}

	//Clean Up CPU memory and close up shop
	free (E);
	free (R);
    
    
	return 0;
}
