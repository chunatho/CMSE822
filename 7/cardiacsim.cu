/* 
 * Solves the Panfilov model using an explicit numerical scheme.
 * Based on code orginally provided by Xing Cai, Simula Research Laboratory 
 * and reimplementation by Scott B. Baden, UCSD
 * 
 * Modified and  restructured by Didem Unat, Koc University
 *
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>
#include <stdlib.h>
#include <iostream>
#include <iomanip>
#include <string.h>
#include <math.h>
#include <sys/time.h>
using namespace std;

# define IMAT(j,i) ( (i) + (n+2)*(j) ) 

// Utilities
// 

// Timer
// Make successive calls and take a difference to get the elapsed time.
static const double kMicro = 1.0e-6;
double getTime() {
	struct timeval TV;
    struct timezone TZ;

    const int RC = gettimeofday(&TV, &TZ);
    if (RC == -1) {
            cerr << "ERROR: Bad call to gettimeofday" << endl;
            return(-1);
    }

    return ( ( (double) TV.tv_sec ) + kMicro*( (double) TV.tv_usec) );

}  // end getTime()

// Allocate a 2D array
double ** alloc2D(int n) {

	// Comments here would be nice
	double **E;

	E = (double**)malloc(sizeof(double*)*n + sizeof(double)*n*n);
	assert(E);

	for( int j = 0; j < n; j++ ) 
		E[j] = (double*)(E+n) + j*n;
	return(E);
}
    
// Reports statistics about the computation
// These values should not vary (except to within roundoff)
// when we use different numbers of  processes to solve the problem
double stats(double *E, int n, double *_mx) {
	double mx = -1;
	double l2norm = 0;

	for ( int j = 1; j <= n; j++ ) 
		for ( int i = 1; i <= n; i++ ) {

			l2norm += E[ IMAT(j,i) ]*E[ IMAT(j,i) ];
			if ( E[ IMAT(j,i) ] > mx )
				mx = E[ IMAT(j,i) ];

		}	

	*_mx = mx;
	l2norm /= ( (double) (n*n) );
	l2norm = sqrt(l2norm);

	return l2norm;
 }

// External functions
extern "C" {
    void splot(double *E, double T, int niter, int m, int n);
}
void cmdLine(int argc, char *argv[], double& T, int& n, int& px, int& py, int& plot_freq, int& no_comm, int&num_threads);


__global__
void first_kernel(double* E_prev, const int n)
{
    int j = blockDim.x * blockIdx.x + threadIdx.x;
    if(j<=n) {
        E_prev[ IMAT(j,0)  ] = E_prev[ IMAT (j,2) ];

		E_prev[ IMAT(j,n+1) ] = E_prev[ IMAT(j,n-1) ];

		E_prev[ IMAT(0,j) ] = E_prev[ IMAT(2,j) ];

		E_prev[ IMAT(n+1,j) ] = E_prev[ IMAT(n-1,j) ];
    }
}
__global__
void second_kernel(double* E, double* E_prev, const int n, const double alpha)
{
    int i = threadIdx.x;
    int j = blockIdx.x;
    int k = blockDim.x * blockIdx.x + threadIdx.x;
    if(k<=(n+2)*(n+2)) {
			E[ IMAT(j,i) ] = E_prev[ IMAT(j,i) ]+alpha*(E_prev[ IMAT(j,i+1) ]
					  + E_prev[ IMAT(j,i-1) ]-4*E_prev[ IMAT(j,i) ]
					  + E_prev[ IMAT(j+1,i) ]+E_prev[ IMAT(j-1,i) ]);
    }
}

__global__
void third_kernel(double* E, double* R, const int n, const double kk, const double dt, const double a)
{
    int i = threadIdx.x;
    int j = blockIdx.x;
    int k = blockDim.x * blockIdx.x + threadIdx.x;
    if(k<=(n+2)*(n+2)) {
			E[ IMAT(j,i) ] = E[ IMAT(j,i) ] - dt*(kk*E[ IMAT(j,i) ]*( E[ IMAT(j,i) ] - a )*( E[ IMAT(j,i) ] - 1 ) 
                + E[ IMAT(j,i) ]*R[ IMAT(j,i) ] );
    }
}
__global__
void fourth_kernel(double* E, double* R, const int n, const double kk, const double dt, const double epsilon, 
        const double M1, const double M2, const double b)
{
    int i = threadIdx.x;
    int j = blockIdx.x;
    int k = blockDim.x * blockIdx.x + threadIdx.x;
    if(k<=(n+2)*(n+2)) {
			R[ IMAT(j,i) ] = R[ IMAT(j,i) ] + dt*( epsilon + M1*R[ IMAT(j,i) ] / ( E[ IMAT(j,i) ] + M2) )*( -R[ IMAT(j,i) ]
                 - kk*E[ IMAT(j,i) ]*(E[ IMAT(j,i) ] - b - 1 ) );
    }
}
void simulate (double* d_E,  double* d_E_prev,double* d_R,
   			   const double alpha, const int n, const double kk,
   			   const double dt, const double a, const double epsilon,
               const double M1,const double  M2, const double b) {
/*
	double * E = (double*)(E_old + n + 2);
	double * E_prev = (double*)(E_prev_old + n + 2);
	double * R = (double*)(R_old + n + 2);
*/
	/* 
	* Copy data from boundary of the computational box 
	* to the padding region, set up for differencing
	* on the boundary of the computational box
	* Using mirror boundaries
	*/
//    printf("ceil((n+2)/256.0): %f \t n: %d \n", ceil((n+2)/256.0), n);
    
    first_kernel<<<ceil((n+2)),n+2>>>(d_E_prev,n);
    second_kernel<<<ceil(n+2),n+2>>>(d_E,d_E_prev,n,alpha);
    third_kernel<<<ceil(n+2),n+2>>>(d_E,d_R,n,kk,dt,a);
    fourth_kernel<<<ceil(n+2),n+2>>>(d_E,d_R,n,kk,dt,epsilon,M1,M2,b);
    

/*
	for ( int j = 1; j <= n; j++ ){ 
		E_prev[ IMAT(j,0) ] = E_prev[ IMAT(j,2) ];
        if(j==27) printf("j: %d \t h_E_prev: %f \t E_prev: %f \n", j, h_E_prev[j], E_prev[j]);
    }
	for ( int j = 1; j <= n; j++ ) 
		E_prev[ IMAT(j,n+1) ] = E_prev[ IMAT(j,n-1) ];

	for ( int i = 1; i <= n; i++ ) 
		E_prev[ IMAT(0,i) ] = E_prev[ IMAT(2,i) ];

	for ( int i = 1; i <=n ; i++ ) 
		E_prev[ IMAT(n+1,i) ] = E_prev[ IMAT(n-1,i) ];
*/
/*
	// Solve for the excitation, the PDE
	for ( int j = 1; j <= n; j++ ){
		for ( int i = 1; i <= n; i++ ) {
			E[ IMAT(j,i) ] = E_prev[ IMAT(j,i) ]+alpha*(E_prev[ IMAT(j,i+1) ]
					  + E_prev[ IMAT(j,i-1) ]-4*E_prev[ IMAT(j,i) ]
					  + E_prev[ IMAT(j+1,i) ]+E_prev[ IMAT(j-1,i) ]);
		}	
}
*/
	/* 
	* Solve the ODE, advancing excitation and recovery to the
	* next timtestep
	*/
/*
	for ( int j = 1; j <= n; j++ ) {
		for ( int i=1; i<=n; i++)
			E[ IMAT(j,i) ] = E[ IMAT(j,i) ] - dt*(kk*E[ IMAT(j,i) ]*( E[ IMAT(j,i) ] - a )*( E[ IMAT(j,i) ] - 1 ) + E[ IMAT(j,i) ]*R[ IMAT(j,i) ] );
	}
*/
/*
	for ( int j = 1; j <= n; j++ ) {
		for ( int i = 1; i <= n; i++ )
			R[ IMAT(j,i) ] = R[ IMAT(j,i) ] + dt*( epsilon + M1*R[ IMAT(j,i) ] / ( E[ IMAT(j,i) ] + M2) )*( -R[ IMAT(j,i) ] - kk*E[ IMAT(j,i) ]*(E[ IMAT(j,i) ] - b - 1 ) );
	}	
*/
}

// Main program
int main (int argc, char** argv) {

	/*
	*  Solution arrays
	*   E is the "Excitation" variable, a voltage
	*   R is the "Recovery" variable
	*   E_prev_old is the Excitation variable for the previous timestep,
	*      and is used in time integration
	*/
//	double **E_old, **R_old, **E_prev_old;

	// Various constants - these definitions shouldn't change
	const double a = 0.1, b = 0.1, kk = 8.0, M1 = 0.07, M2 = 0.3, epsilon = 0.01, d = 5e-5;

	double T = 1000.0;
	int n = 200;
	int plot_freq = 0;
	int px = 1, py = 1;
	int no_comm = 0;
	int num_threads=1; 

	cmdLine( argc, argv, T, n,px, py, plot_freq, no_comm, num_threads);

	// Allocate contiguous memory for solution arrays
	// The computational box is defined on [1:m+1,1:n+1]
	// We pad the arrays in order to facilitate differencing on the 
	// boundaries of the computation box
//	E_old = alloc2D( n + 2 );
//	E_prev_old = alloc2D( n + 2 );
//	R_old = alloc2D( n + 2 );

    double *E; //= (double*)(E_old + n + 2);
    double *E_prev; // = (double*)(E_prev_old + n + 2);
    double *R; // = (double*)(R_old + n + 2);
	
    E = (double *)malloc((n+2)*(n+2)*sizeof(double));
    E_prev = (double *)malloc((n+2)*(n+2)*sizeof(double));
    R = (double *)malloc((n+2)*(n+2)*sizeof(double));

    // Initialization
	for ( int j = 1; j <= n; j++ )
		for ( int i = 1; i <= n; i++ )
			E_prev[ IMAT(j,i) ] = R[ IMAT(j,i) ] = 0;

	for ( int j = 1; j <= n; j++ )
		for ( int i = n/2 + 1; i <= n; i++ )
			E_prev[ IMAT(j,i) ] = 1.0;

	for ( int j = n/2 + 1; j <= n; j++ )
		for ( int i = 1; i <= n; i++ )
			R[ IMAT(j,i) ] = 1.0;

	double dx = 1.0/n;

	// For time integration, these values shouldn't change 
	double rp= kk*( b + 1 )*( b + 1 ) / 4;
	double dte = ( dx*dx ) / ( d*4 +  ( dx*dx ) * ( rp + kk ) );
	double dtr = 1 / ( epsilon + (M1/M2)*rp );
	double dt = ( dte < dtr ) ? 0.95*dte : 0.95* dtr;
	double alpha = d*dt / ( dx*dx );
/*
	cout << "Grid Size       : " << n << endl; 
	cout << "Duration of Sim : " << T << endl; 
	cout << "Time step dt    : " << dt << endl; 
	cout << "Process geometry: " << px << " x " << py << endl;
	if (no_comm)
		cout << "Communication   : DISABLED" << endl;

	cout << endl;
*/
	// Start the timer
	double t0 = getTime();


	// Simulated time is different from the integer timestep number

	// Simulated time
	double t = 0.0;

	// Integer timestep number
	int niter=0;
    double *d_E_prev, *d_E, *d_R;
    hipMalloc(&d_E,(n+2)*(n+2)*sizeof(double));
    hipMalloc(&d_E_prev,(n+2)*(n+2)*sizeof(double));
    hipMalloc(&d_R,(n+2)*(n+2)*sizeof(double));
    hipMemcpy(d_E, E, (n+2)*(n+2)*sizeof(double), hipMemcpyHostToDevice); 
    hipMemcpy(d_E_prev, E_prev, (n+2)*(n+2)*sizeof(double), hipMemcpyHostToDevice); 
    hipMemcpy(d_R, R, (n+2)*(n+2)*sizeof(double), hipMemcpyHostToDevice); 

	while ( t < T ) {
	
		// Don't know what the fuck this is supposed to do since there's no comment
		t += dt;
		niter++;


		simulate(d_E, d_E_prev, d_R, alpha, n, kk, dt, a, epsilon, M1, M2, b); 

		//swap current E with previous E
		double *tmp = d_E; d_E = d_E_prev; d_E_prev = tmp;

		if ( plot_freq ) {
			int k = ( (int) (t / plot_freq) );
			if ( (t - k*plot_freq ) < dt ) {
				splot(d_E,t,niter,n+2,n+2);
			}
		}
	} //end of while loop
    hipMemcpy(E, d_E, (n+2)*(n+2)*sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(E_prev, d_E_prev, (n+2)*(n+2)*sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(R, d_R, (n+2)*(n+2)*sizeof(double), hipMemcpyDeviceToHost);
    hipFree(d_E);
    hipFree(d_E_prev);
    hipFree(d_R);

	double time_elapsed = getTime() - t0;

	double Gflops = (double)(niter * (1E-9 * n * n ) * 28.0) / time_elapsed;
	double BW = (double)(niter * 1E-9 * (n * n * sizeof(double) * 4.0  ))/time_elapsed;
/*
	cout << "Number of Iterations        : " << niter << endl;
	cout << "Elapsed Time (sec)          : " << time_elapsed << endl;
	cout << "Sustained Gflops Rate       : " << Gflops << endl; 
	cout << "Sustained Bandwidth (GB/sec): " << BW << endl << endl; 
*/
	double mx;
	double l2norm = stats(E_prev,n,&mx);
//	cout << "Max: " << mx <<  " L2norm: "<< l2norm << endl;
    printf("v1 \t %d \t %.10e \t %.10e \t %.6e \n",n,mx,l2norm,time_elapsed);
    if (plot_freq)	{	
		cout << "\n\nEnter any input to close the program and the plot..." << endl;
		getchar();
	}

/*
	free (E_old);
	free (E_prev_old);
	free (R_old);
*/ 
       
	free (E);
	free (E_prev);
	free (R);
	
    return 0;
}
